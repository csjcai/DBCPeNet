#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/WBCprior_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void DCPForward(const int nthreads, const int channels,
    const Dtype* const bottom_data, const int height, const int width, 
    const int pooled_height, const int pooled_width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, Dtype* const top_data, int* mask) {
  
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int n = index / pooled_width / pooled_height;  
    int hstart = ph - pad_h;
    int wstart = pw - pad_w;
    const int hend = min(hstart + kernel_h, height);
    const int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype minval = FLT_MAX;
    int minidx = -1;
    const Dtype* const bottom_slice = bottom_data + (n * channels * height * width);
    for (int c = 0; c < channels; ++c) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          if (bottom_slice[c * height * width + h * width + w] < minval) {
            minidx = c * height * width + h * width + w;
            minval = bottom_slice[minidx];
          }
        }
      }
    }
    top_data[index] = minval;
    mask[index] = minidx;
  }
}

template <typename Dtype>
__global__ void WCPForward(const int nthreads, const int channels,
    const Dtype* const bottom_data, const int height, const int width, 
    const int pooled_height, const int pooled_width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, Dtype* const top_data, int* mask) {
  
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int n = index / pooled_width / pooled_height;  
    int hstart = ph - pad_h;
    int wstart = pw - pad_w;
    const int hend = min(hstart + kernel_h, height);
    const int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    const Dtype* const bottom_slice = bottom_data + (n * channels * height * width);
    for (int c = 0; c < channels; ++c) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          if (bottom_slice[c * height * width + h * width + w] > maxval) {
            maxidx = c * height * width + h * width + w;
            maxval = bottom_slice[maxidx];
          }
        }
      }
    }
    top_data[index] = maxval;
    mask[index] = maxidx;
  }
}


template <typename Dtype>
void WbcpriorLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();

  int* mask = NULL;

  switch (this->layer_param_.wbcprior_param().typeprior()) {
  case WbcPriorParameter_PriorMethod_DARK:
    mask = black_mask_.mutable_gpu_data();
    DCPForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, channels_, bottom_data, height_, width_, pooled_height_, pooled_width_, 
        kernel_h_, kernel_w_, pad_h_, pad_w_, top_data, mask);
    break;
  case WbcPriorParameter_PriorMethod_WHITE:
    mask = white_mask_.mutable_gpu_data();
    WCPForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, channels_, bottom_data, height_, width_, pooled_height_, pooled_width_, 
        kernel_h_, kernel_w_, pad_h_, pad_w_, top_data, mask);
    break;
  default:
    LOG(FATAL) << "Unknown prior method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void DCPBackward(const int nthreads, const Dtype* const top_diff,
    const int* const mask, const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, const int kernel_h, const int kernel_w, 
    const int pad_h, const int pad_w, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart =
         (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) + 1;
    const int phend = min((h + pad_h) + 1, pooled_height);
    const int pwstart =
         (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) + 1;
    const int pwend = min((w + pad_w) + 1, pooled_width);
    
    Dtype gradient = 0;
    const int offset = n * pooled_height * pooled_width;
    const Dtype* const top_diff_slice = top_diff + offset;
    const int* const mask_slice = mask + offset;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (mask_slice[ph * pooled_width + pw] == c * height * width + h * width + w) {
           gradient += top_diff_slice[ph * pooled_width + pw];
         }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
__global__ void WCPBackward(const int nthreads, const Dtype* const top_diff,
    const int* const mask, const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, const int kernel_h, const int kernel_w, 
    const int pad_h, const int pad_w, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart =
         (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) + 1;
    const int phend = min((h + pad_h) + 1, pooled_height);
    const int pwstart =
         (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) + 1;
    const int pwend = min((w + pad_w) + 1, pooled_width);
    
    Dtype gradient = 0;
    const int offset = n * pooled_height * pooled_width;
    const Dtype* const top_diff_slice = top_diff + offset;
    const int* const mask_slice = mask + offset;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (mask_slice[ph * pooled_width + pw] == c * height * width + h * width + w) {
           gradient += top_diff_slice[ph * pooled_width + pw];
         }
      }
    }
    bottom_diff[index] = gradient;
  }
}


template <typename Dtype>
void WbcpriorLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);

  const int* mask = NULL;

  switch (this->layer_param_.wbcprior_param().typeprior()) {
  case WbcPriorParameter_PriorMethod_DARK:
    mask = black_mask_.gpu_data();
    DCPBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, channels_, height_, width_, pooled_height_, pooled_width_,
        kernel_h_, kernel_w_, pad_h_, pad_w_, bottom_diff);
    break;
  case WbcPriorParameter_PriorMethod_WHITE:
    mask = white_mask_.gpu_data();
    WCPBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, channels_, height_, width_, pooled_height_, pooled_width_,
        kernel_h_, kernel_w_, pad_h_, pad_w_, bottom_diff);
    break;
  default:
    LOG(FATAL) << "Unknown prior method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(WbcpriorLayer);


}  // namespace caffe
